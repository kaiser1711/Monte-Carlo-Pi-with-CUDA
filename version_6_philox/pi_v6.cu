#include <iostream>
#include <limits>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <chrono>
#include <fstream>
#include <random>
#include <cstring>


using namespace std;

typedef unsigned long long Count;

const int seq_iter = 100000 ;

const Count N_BLOCKS = 1024;
const Count N_THREADS = 1024;
const Count WARP_SIZE = 32;
const Count MAX_WARPS_PER_BLOCK = N_THREADS/WARP_SIZE;

// This kernel is 
__global__ void picount(Count *totals, int seq_iter) {
	// Define some shared memory: all threads in this block
	__shared__ Count counter[N_THREADS];

	// Unique ID of the thread
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	// Initialize RNG
	hiprandStatePhilox4_32_10_t rng;
	hiprand_init(clock64(), tid, 0, &rng);

	// Initialize the counter
	counter[threadIdx.x] = 0;

	// Computation loop
	for (int i = 0; i < seq_iter; i++) {
		float x = hiprand_uniform(&rng); // Random x position in [0,1]
		float y = hiprand_uniform(&rng); // Random y position in [0,1]
		counter[threadIdx.x] += 1 - int(x * x + y * y); // Hit test
	}

	__syncthreads();

   // Warp-level reduction
    unsigned int lane = threadIdx.x % WARP_SIZE;
    unsigned int warp_id = threadIdx.x / WARP_SIZE;
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        counter[threadIdx.x] += __shfl_down_sync(0xffffffff, counter[threadIdx.x], offset, WARP_SIZE);
    }

    // The first thread in each warp should sum the warp results
    __shared__ Count warp_totals[MAX_WARPS_PER_BLOCK];
    if (lane == 0) {
        warp_totals[warp_id] = counter[threadIdx.x];
    }
    __syncthreads();

    // The first thread in the block should sum the warp totals
    if (threadIdx.x == 0) {
        totals[blockIdx.x] = 0;
        for (int i = 0; i < MAX_WARPS_PER_BLOCK; i++) {
            totals[blockIdx.x] += warp_totals[i];
        }
    }
}

int main(int argc, char **argv) {

	// Allocate host and device memory to store the counters
	Count *hOut, *dOut;
	hOut = new Count[N_BLOCKS]; // Host memory
	hipMalloc(&dOut, sizeof(Count)*N_BLOCKS); // Device memory

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start time
    hipEventRecord(start, 0);


	// Launch kernel
	picount<<<N_BLOCKS,N_THREADS>>>(dOut,seq_iter);


    // Record the stop time
    hipEventRecord(stop, 0);

    // Synchronize and measure the elapsed time
    hipEventSynchronize(stop);
    float kernel_time_ms;
    hipEventElapsedTime(&kernel_time_ms, start, stop);

    std::cout << "Kernel execution time: " << kernel_time_ms << " ms" << std::endl;

    // Clean up CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);


	// Copy back memory used on device and free
	hipMemcpy(hOut, dOut, sizeof(Count)*N_BLOCKS,hipMemcpyDeviceToHost);
	hipFree(dOut);

	// Compute total hits
	Count total = 0;
	for(int i=0; i<N_BLOCKS; i++)
	{
		total += hOut[i];
	}

	Count tests = seq_iter*N_BLOCKS*N_THREADS;
	cout << "Approximated PI using " << tests << " random tests\n";

	// Set maximum precision for decimal printing
	cout.precision(std::numeric_limits<double>::max_digits10);
	cout << "PI ~= " << 4.0 * (double)total/(double)tests << endl;
	cout << "Tests per ms: " << tests/kernel_time_ms << endl;

	return 0;
}
