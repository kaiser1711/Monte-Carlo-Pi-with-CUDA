#include <iostream>
#include <limits>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <chrono>
#include <fstream>
#include <random>
#include <cstring>


using namespace std;

typedef unsigned long long Count;

const int seq_iter = 10000 ;

const Count N_BLOCKS = 1024;
const Count N_THREADS = 1024;
const Count WARP_SIZE = 32;

// This kernel is 
__global__ void picount(Count *totals, int seq_iter) {
	// Define some shared memory: all threads in this block
    __shared__ Count shared_counter;

    // Unique ID of the thread
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Initialize RNG
    hiprandState_t rng;
    hiprand_init(clock64(), tid, 0, &rng);

    // Initialize the shared counter
    if (threadIdx.x == 0) {
        shared_counter = 0;
    }
    __syncthreads();

    // Computation loop
    for (int i = 0; i < seq_iter; i++) {
        float x = hiprand_uniform(&rng); // Random x position in [0,1]
        float y = hiprand_uniform(&rng); // Random y position in [0,1]
        bool hit = (x * x + y * y <= 1.0f); // Hit test
        atomicAdd(&shared_counter, hit ? 1 : 0); // Update shared counter
    }
    __syncthreads();

    // The first thread in the block should store the result
    if (threadIdx.x == 0) {
        totals[blockIdx.x] = shared_counter;
    }
}

int main(int argc, char **argv) {


	// Allocate host and device memory to store the counters
	Count *hOut, *dOut;
	hOut = new Count[N_BLOCKS]; // Host memory
	hipMalloc(&dOut, sizeof(Count)*N_BLOCKS); // Device memory

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start time
    hipEventRecord(start, 0);


	// Launch kernel
	picount<<<N_BLOCKS,N_THREADS>>>(dOut,seq_iter);


    // Record the stop time
    hipEventRecord(stop, 0);

    // Synchronize and measure the elapsed time
    hipEventSynchronize(stop);
    float kernel_time_ms;
    hipEventElapsedTime(&kernel_time_ms, start, stop);

    std::cout << "Kernel execution time: " << kernel_time_ms << " ms" << std::endl;

    // Clean up CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);


	// Copy back memory used on device and free
	hipMemcpy(hOut, dOut, sizeof(Count)*N_BLOCKS,hipMemcpyDeviceToHost);
	hipFree(dOut);

	// Compute total hits
	Count total = 0;
	for(int i=0; i<N_BLOCKS; i++)
	{
		total += hOut[i];
	}

	Count tests = seq_iter*N_BLOCKS*N_THREADS;
	cout << "Approximated PI using " << tests << " random tests\n";

	// Set maximum precision for decimal printing
	cout.precision(std::numeric_limits<double>::max_digits10);
	cout << "PI ~= " << 4.0 * (double)total/(double)tests << endl;
	cout << "Tests per ms: " << tests/kernel_time_ms << endl;


	return 0;
}
